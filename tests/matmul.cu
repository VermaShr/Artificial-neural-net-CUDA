#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <hip/hip_runtime.h>

#define NUM_THREADS 256
#define EPSILON 0.0001
//#define EPSILON 0.00001 --> error is too small

// KERNEL: x*A = B
__global__ void MatMul(float* x, float* A, float* B, int height, int width)
{
    // index into flattened weights matrix
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // index into the input vector
    int row = i / width;

    // index into the output vector
    int col = i % width;

    // zero out resultant vector B
    if (i < width) B[i] = 0.0;

    __syncthreads();

    if ((i < height * width) && (row < height))
    {
        // TODO: atomicAdd to local, shared output vectors --> atomicAdd to global
        atomicAdd(&B[col], x[row] * A[i]);
    }
}

// HOST
int main(int argc, char** argv)
{
    // Variables
    float *h_x, *h_A, *h_B, *d_x, *d_A, *d_B;
    int height = 256;
    int width = 10;

    // Allocate vectors and matrices in host memory and device memory
    h_x = (float*)malloc(height*sizeof(float));
    h_A = (float*)malloc(height*width*sizeof(float));
    h_B = (float*)malloc(width*sizeof(float));
    hipMalloc((void**)&d_x, height*sizeof(float));
    hipMalloc((void**)&d_A, height*width*sizeof(float));
    hipMalloc((void**)&d_B, width*sizeof(float));

    // Initialize input vector x
    for (int i = 0; i < height; ++i)
    {
        h_x[i] = rand() / (float)RAND_MAX;
    }

    // Initialize input matrix A
    for (int i = 0; i < height; i++)
    {
        for (int j = 0; j < width; j++)
        {
            h_A[i*width + j] = rand() / (float)RAND_MAX;
        }
    }

    // Copy vectors from host memory to device memory
    hipMemcpy(d_x, h_x, height*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_A, h_A, height*width*sizeof(float), hipMemcpyHostToDevice);

    // FILL IN KERNEL SETUP AND INVOCATION

    int blocks = (height*width) / NUM_THREADS;
    if ((height*width) % NUM_THREADS != 0) blocks++;

    MatMul <<< blocks, NUM_THREADS  >>> (d_x, d_A, d_B, height, width);

    hipDeviceSynchronize();

    // Copy result from device memory to host memory
    hipMemcpy(h_B, d_B, width*sizeof(float), hipMemcpyDeviceToHost);

    bool correct = true;

    // Calculate solution on the host and compare
    float* result = (float*)malloc(width*sizeof(float));

    for (int i = 0; i < height; i++)
    {
        for (int j = 0; j < width; j++)
        {
            // zero out result elements
            if (i == 0) result[j] = 0.0;

            result[j] += h_x[i] * h_A[i*width + j];
        }
    }

    for (int j = 0; j < width; j++)
    {
        if (fabs(h_B[j] - result[j]) > EPSILON)
        {
            printf("ERROR: expected h_B[%i] = %f but received %f\n", j, result[j], h_B[j]);
            correct = false;
            break;
        }
    }

    if (correct) printf("---PASSED---\n");

    // Free host and device memory
    hipFree(d_x); hipFree(d_A); hipFree(d_B);
    free(h_x); free(h_A); free(h_B); free(result);
}
