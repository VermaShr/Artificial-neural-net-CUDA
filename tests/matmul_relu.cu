#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <hip/hip_runtime.h>

#define NUM_THREADS 1024
//#define EPSILON 0.0001
#define EPSILON 0.00001 //--> error is too small

#define height 256
#define width 10

#define UPPER 0.01
#define LOWER -0.01

// KERNEL: x*A = B
__global__ void MatMul(float* x, float* A, float* B)
{
    // index into flattened weights matrix
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // index into the input vector
    int row = i / width;

    // index into the output vector
    int col = i % width;

    // zero out resultant vector B
    if (i < width) B[i] = 0.0;

    __syncthreads();

    if ((i < height * width) && (row < height))
    {
        // TODO: atomicAdd to local, shared output vectors --> atomicAdd to global
        atomicAdd(&B[col], x[row] * A[i]);
        __syncthreads();

        if (i < width && B[i] < 0.0) B[i] = 0.0;
    }
}

// HOST
int main(int argc, char** argv)
{
    // Variables
    float *h_x, *h_A, *h_B, *d_x, *d_A, *d_B;

    // Allocate vectors and matrices in host memory and device memory
    h_x = (float*)malloc(height*sizeof(float));
    h_A = (float*)malloc(height*width*sizeof(float));
    h_B = (float*)malloc(width*sizeof(float));
    hipMalloc((void**)&d_x, height*sizeof(float));
    hipMalloc((void**)&d_A, height*width*sizeof(float));
    hipMalloc((void**)&d_B, width*sizeof(float));

    // Initialize input vector x
    for (int i = 0; i < height; ++i)
    {
        h_x[i] = rand() / (float)RAND_MAX - 0.5;
    }

    // Initialize input matrix A
    for (int i = 0; i < height; i++)
    {
        for (int j = 0; j < width; j++)
        {
            // initialize weights matrix values to be between LOWER and UPPER
            h_A[i*width + j] = (rand() / (float)RAND_MAX)*(UPPER - LOWER) + LOWER;
        }
    }

    // Copy vectors from host memory to device memory
    hipMemcpy(d_x, h_x, height*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_A, h_A, height*width*sizeof(float), hipMemcpyHostToDevice);

    // FILL IN KERNEL SETUP AND INVOCATION

    int blocks = (height*width) / NUM_THREADS;
    if ((height*width) % NUM_THREADS != 0) blocks++;

    MatMul <<< blocks, NUM_THREADS  >>> (d_x, d_A, d_B);

    hipDeviceSynchronize();

    // Copy result from device memory to host memory
    hipMemcpy(h_B, d_B, width*sizeof(float), hipMemcpyDeviceToHost);

    bool correct = true;

    // Calculate solution on the host and compare
    float* result = (float*)malloc(width*sizeof(float));

    for (int i = 0; i < height; i++)
    {
        for (int j = 0; j < width; j++)
        {
            // zero out result elements
            if (i == 0) result[j] = 0.0;

            result[j] += h_x[i] * h_A[i*width + j];
        }
    }

    for (int j = 0; j < width; j++)
    {
        if (result[j] < 0.0)
        {
            printf("--old result[%i]: %f--", j, result[j]);
            result[j] = 0.0;
        }

        if (fabs(h_B[j] - result[j]) > EPSILON)
        {
            printf("ERROR: expected h_B[%i] = %f but received %f\n", j, result[j], h_B[j]);
            correct = false;
            //break;
        }
        else
        {
            printf("result[j]: %f\th_B[j]: %f\n", result[j], h_B[j]);
        }        
    }

    if (correct) printf("---PASSED---\n");

    // Free host and device memory
    hipFree(d_x); hipFree(d_A); hipFree(d_B);
    free(h_x); free(h_A); free(h_B); free(result);
}
