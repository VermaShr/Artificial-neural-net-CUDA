#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <hip/hip_runtime.h>

#define NUM_THREADS 1024
#define EPSILON 0.0001
//#define EPSILON 0.00001 //--> sometimes fails from too small error

#define height 256
#define width 10

#define UPPER 0.01
#define LOWER -0.01

// KERNEL: x*A = B
__global__ void MatMul(float* x, float* A, float* B)
{
    // index into flattened weights matrix
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // index into the input vector
    int row = i / width;

    // index into the output vector
    int col = i % width;

    //__shared__ float local_output[];

    // zero out resultant vector B
    if (i < width) B[i] = 0.0;

    __syncthreads();

    if ((i < height * width) && (row < height))
    {
        // TODO: atomicAdd to local, shared output vectors --> atomicAdd to global
        atomicAdd(&B[col], x[row] * A[i]);
        __syncthreads();

        if (i < width)
        {
            // SOFTMAX CALCULATION
            __shared__ float sum;

            // 1. store the exp() of each output value
            __shared__ float exp_vector[width];
            exp_vector[i] = expf(B[i]);
           
            // 2. calculate the sum of all the exponent values
            //  --> width < BLOCK_SIZE, so this will only be in the first block
            if (threadIdx.x == 0) sum = 0.0;
            __syncthreads(); // wait for sum to ve zeroed

            atomicAdd(&sum, exp_vector[i]);
            __syncthreads();

            // 3. store new output value
            B[i] = exp_vector[i] / sum;
        }
    }
}

// HOST
int main(int argc, char** argv)
{
    // Variables
    float *h_x, *h_A, *h_B, *d_x, *d_A, *d_B;
    //int height = 256;
    //int width = 100;

    // Allocate vectors and matrices in host memory and device memory
    h_x = (float*)malloc(height*sizeof(float));
    h_A = (float*)malloc(height*width*sizeof(float));
    h_B = (float*)malloc(width*sizeof(float));
    hipMalloc((void**)&d_x, height*sizeof(float));
    hipMalloc((void**)&d_A, height*width*sizeof(float));
    hipMalloc((void**)&d_B, width*sizeof(float));

    // Initialize input vector x
    for (int i = 0; i < height; ++i)
    {
        //h_x[i] = (rand() / (float)RAND_MAX)*(UPPER - LOWER) + LOWER;
        h_x[i] = rand() / (float)RAND_MAX - 0.5;
        //printf("h_x[%i]: %f\n", i, h_x[i]);
    }

    // Initialize input matrix A
    for (int i = 0; i < height; i++)
    {
        for (int j = 0; j < width; j++)
        {
            // initialize weights matrix values to be between LOWER and UPPER
            h_A[i*width + j] = (rand() / (float)RAND_MAX)*(UPPER - LOWER) + LOWER;
        }
    }

    // Copy vectors from host memory to device memory
    hipMemcpy(d_x, h_x, height*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_A, h_A, height*width*sizeof(float), hipMemcpyHostToDevice);

    // FILL IN KERNEL SETUP AND INVOCATION

    int blocks = (height*width) / NUM_THREADS;
    if ((height*width) % NUM_THREADS != 0) blocks++;

    MatMul <<< blocks, NUM_THREADS  >>> (d_x, d_A, d_B);

    hipDeviceSynchronize();

    // Copy result from device memory to host memory
    hipMemcpy(h_B, d_B, width*sizeof(float), hipMemcpyDeviceToHost);

    bool correct = true;

    // Calculate solution on the host and compare
    float* result = (float*)malloc(width*sizeof(float));

    for (int i = 0; i < height; i++)
    {
        for (int j = 0; j < width; j++)
        {
            // zero out result elements
            if (i == 0) result[j] = 0.0;

            result[j] += h_x[i] * h_A[i*width + j];
        }
    }

    // 1. calculate sum
    float sum = 0.0;
    float exp_vec[width];
    for (int j = 0; j < width; j++)
    {
        //exp_vec[j] = (float)exp((double)result[j]);
        exp_vec[j] = expf(result[j]);

        // sum up the exp value just calculated
        sum += exp_vec[j];
        //printf("result[%i]: %f\n", j, result[j]);
    }

    //printf("-->result sum: %f\n", sum);
    
    float r_sum = 0.0;
    float b_sum = 0.0;

    for (int j = 0; j < width; j++)
    {
        result[j] = exp_vec[j] / sum;
        
        r_sum += result[j];
        b_sum += h_B[j];

        if (fabs(h_B[j] - result[j]) > EPSILON)
        {
            printf("ERROR: expected h_B[%i] = %f but received %f\n", j, result[j], h_B[j]);
            correct = false;
            //break;
        }
        else
        {
            printf("result[j]: %f\th_B[j]: %f\n", result[j], h_B[j]);
        }        
    }

    printf("-->result sum: %f\n", r_sum);
    printf("-->h_B sum: %f\n", b_sum);
    
    if (correct) printf("\n---PASSED---\n");
    else printf("\n---FAILED---\n");

    // Free host and device memory
    hipFree(d_x); hipFree(d_A); hipFree(d_B);
    free(h_x); free(h_A); free(h_B); free(result);
}
